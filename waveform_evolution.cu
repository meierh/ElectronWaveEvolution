#include "hip/hip_runtime.h"
#include <waveform_evolution.hpp>

#include <iostream>

__global__ void check_collision_kernel
(
	const std::uint64_t* wave_data,
	std::uint64_t wave_data_len,
	std::uint64_t activation,
	std::uint64_t deactivation,
	bool* collision,
	std::uint64_t* non_collision_offset
)
{
	// TODO: Compute of collision occures and set bit in collision_bits if it happens
	std::uint64_t wave_data_index = blockDim.x*blockIdx.x + threadIdx.x;
	if(wave_data_index<wave_data_len)
	{
		std::uint64_t wave = wave_data[wave_data_index];
		bool col = (bool)((wave & activation) | ((~wave) & deactivation));
		collision[wave_data_index] = col;
		non_collision_offset[wave_data_index] = col ? 0 : 1;
	}
}

template<uint num_threads>
concept ThreadsOK = num_threads%8==0 && num_threads>0 && num_threads<30000;

template<uint num_threads>
__global__ void inclusive_scan_kernel
(
	std::uint64_t* non_collision_offset,
	std::uint64_t wave_data_len,
	std::uint64_t* non_collision_offset_endBlock,
	std::uint16_t blockOffset
)
requires ThreadsOK<num_threads>
{
	std::uint64_t wave_data_index = blockDim.x*(blockIdx.x+blockOffset) + threadIdx.x;
	__shared__ std::uint64_t offsets[num_threads];
	offsets[threadIdx.x] = 0;
	if(wave_data_index<wave_data_len)
	{
		offsets[threadIdx.x] = non_collision_offset[wave_data_index];
	}
	if(threadIdx.x==0 && blockIdx.x>blockOffset)
	{
		offsets[threadIdx.x] += non_collision_offset_endBlock[blockIdx.x];
	}
	__syncthreads();

	for(uint stride = 1; stride<=blockDim.x; stride*=2)
	{
		__syncthreads();
		uint indx = (threadIdx.x+1)*2*stride-1;
		if(indx<num_threads)
			offsets[indx] += offsets[indx-stride];
	}
	for(int stride = num_threads/4; stride>0; stride/=2)
	{
		__syncthreads();
		uint indx = (threadIdx.x+1)*2*stride-1;
		if(indx+stride<num_threads)
			offsets[indx+stride] += offsets[indx];
	}
	__syncthreads();

	if(wave_data_index<wave_data_len)
	{
		non_collision_offset[wave_data_index] = offsets[threadIdx.x];
	}

	if(threadIdx.x==blockDim.x-1)
		non_collision_offset_endBlock[blockIdx.x+blockOffset] = offsets[blockDim.x-1];
}

void inclusive_scan
(
	std::uint64_t* non_collision_offset,
	std::uint64_t wave_data_len
)
{
	hipError_t allocError;
	constexpr uint blockSize = 32;
	int gridSize = (wave_data_len/blockSize) + 1;

	pmpp::cuda_ptr<std::uint64_t[]> non_collision_offset_endBlock = pmpp::make_managed_cuda_array<std::uint64_t>(gridSize,hipMemAttachGlobal,&allocError);

	for(std::uint16_t blockOffset=0; blockOffset<gridSize; blockOffset++, gridSize--)
	{
		inclusive_scan_kernel<blockSize><<<dim3(gridSize),dim3(blockSize)>>>
		(
			non_collision_offset,
			wave_data_len,
			non_collision_offset_endBlock.get(),
			blockOffset
		);
	}
}

__global__ void evolve_kernel
(
	const std::uint64_t* wave_data,
	std::uint64_t wave_data_len,
	std::uint64_t activation,
	std::uint64_t deactivation,
	const bool* collision,
	const std::uint64_t* non_collision_offset,
 	std::uint64_t* wave_data_out
)
{
	// TODO: Compute evolved data and store in dst_data according to offsets numbers
	std::uint64_t wave_data_index = blockDim.x*blockIdx.x + threadIdx.x;
	if(wave_data_index < wave_data_len)
	{
		std::uint64_t wave = wave_data[wave_data_index];
		bool wave_collision = collision[wave_data_index];
		std::uint64_t wave_offset = non_collision_offset[wave_data_index];

		if(!wave_collision)
		{
			std::uint64_t new_wave = wave;
			new_wave |= activation;
			new_wave &= ~deactivation;
			wave_data_out[wave_data_len-1+wave_offset] = new_wave;
		}
	}
}

void collisionEvaluation
(
	cuda::std::span<std::uint64_t const> const & device_wavefunction,
	std::uint64_t activation,
	std::uint64_t deactivation,
	pmpp::cuda_ptr<bool[]>& collisions,
	pmpp::cuda_ptr<std::uint64_t[]>& non_collision_offset
)
{
	hipError_t allocError;
	dim3 gridSz;

	std::size_t collision_size = device_wavefunction.size();
	collisions = pmpp::make_managed_cuda_array<bool>(collision_size,hipMemAttachGlobal,&allocError);
	non_collision_offset = pmpp::make_managed_cuda_array<std::uint64_t>(collision_size,hipMemAttachGlobal,&allocError);
	constexpr uint num_threads = 32;
	gridSz = { (static_cast<uint>(device_wavefunction.size())/num_threads)+1 };
	check_collision_kernel<<<gridSz,dim3(num_threads)>>>
	(
		device_wavefunction.data(),
		device_wavefunction.size(),
		activation,
		deactivation,
		collisions.get(),
		non_collision_offset.get()
	);
	hipDeviceSynchronize();
}

void computeOffsets
(
	const cuda::std::span<std::uint64_t const>& device_wavefunction,
	pmpp::cuda_ptr<std::uint64_t[]>& non_collision_offset,
	std::uint64_t& maxOffset
)
{
	inclusive_scan(non_collision_offset.get(),device_wavefunction.size());
	hipMemcpy
	(
		&maxOffset,
		non_collision_offset.get()+device_wavefunction.size()-1,
		sizeof(std::uint64_t),
		hipMemcpyDeviceToHost
	);
}

void evolutionEvaluation
(
	const cuda::std::span<std::uint64_t const> & device_wavefunction,
	std::uint64_t activation,
	std::uint64_t deactivation,
	const pmpp::cuda_ptr<bool[]>& collisions,
	const pmpp::cuda_ptr<std::uint64_t[]>& non_collision_offset,
	std::uint64_t maxOffset,
	cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t>& waveOut
)
{
	hipError_t allocError;
	dim3 gridSz,blockSz;
	pmpp::cuda_ptr<std::uint64_t[]>& wave_data_out = waveOut.first;
	waveOut.second = device_wavefunction.size()+maxOffset;
	wave_data_out = pmpp::make_managed_cuda_array<std::uint64_t>
	(
		device_wavefunction.size()+maxOffset,
		hipMemAttachGlobal,
		&allocError
	);
	blockSz = { 32 };
	gridSz = { 123 };
	evolve_kernel<<<gridSz,blockSz>>>
	(
		device_wavefunction.data(),
		device_wavefunction.size(),
		activation,
		deactivation,
		collisions.get(),
		non_collision_offset.get(),
		wave_data_out.get()
	);
	hipMemcpy
	(
		wave_data_out.get(),
		device_wavefunction.data(),
		device_wavefunction.size()*sizeof(std::uint64_t),
		hipMemcpyDeviceToDevice
	);
	hipDeviceSynchronize();
}

cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> evolve_operator(
	cuda::std::span<std::uint64_t const> device_wavefunction,
	std::uint64_t activation, std::uint64_t deactivation
)
{
	/*
	 * Compute collision data
	 */
	pmpp::cuda_ptr<bool[]> collisions;
	pmpp::cuda_ptr<std::uint64_t[]> non_collision_offset;
	collisionEvaluation(device_wavefunction,activation,deactivation,collisions,non_collision_offset);

	/*
	 * Compute offsets
	 */
	std::uint64_t maxOffset;
	computeOffsets(device_wavefunction,non_collision_offset,maxOffset);

	/*
	 * Compute evolution
	 */
	cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> waveOut;
	evolutionEvaluation
	(
		device_wavefunction,
		activation,
		deactivation,
		collisions,
		non_collision_offset,
		maxOffset,
		waveOut
	);

	return waveOut;
}

cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> evolve_ansatz(
	cuda::std::span<std::uint64_t const> device_wavefunction,
	cuda::std::span<std::uint64_t const> activations,
	cuda::std::span<std::uint64_t const> deactivations
)
{
	/* TODO */
	cuda::std::pair<pmpp::cuda_ptr<std::uint64_t[]>, std::size_t> result;
	for(std::uint64_t operatorInd=0; operatorInd<activations.size(); operatorInd++)
	{
		result = evolve_operator(device_wavefunction,activations[operatorInd],deactivations[operatorInd]);
		device_wavefunction = cuda::std::span<std::uint64_t const>(result.first.get(),result.second);
	}
	return result;
}
